#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <hiprand/hiprand_kernel.h>

#include <time.h>
#include <float.h>
#include <hip/hip_math_constants.h>

#include "vec3.h"
#include "ray.h"
#include "bsdf.h"
#include "sphere.h"
#include "hitable.h"
#include "hitable_list.h"
#include "camera.h"
#include "sampler.h"
#include "light.h"
#include "triangle.h"
#include "utils/OBJ_Loader.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << hipGetErrorString(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // printf("GPU kernel assert: %s %s %d\n", hipGetErrorString(result), file, line);
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ vec3 global_illumination(const ray& r, hitable **world, hiprandState *local_rand_state) {
   ray cur_ray = r;
   vec3 cur_attenuation = vec3(1.0f, 1.0f, 1.0f);
   unit_sphere_sampler sampler;
   vec3 reflectance = vec3(1., 1., 1.);
   vec3 f = reflectance / HIP_PI_F;
   for(int i = 0; i < r.depth; i++) 
   {
      hit_record rec;
      if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) 
      {
        vec3 target;
        double pdf;
        vec3 f = rec.BSDF->evaluate(r.direction(), &target, rec.p, rec.normal, &pdf, local_rand_state);
        cur_attenuation *= f;
        cur_ray = ray(rec.p, target-rec.p);
      }
      else 
      {
            if (i > 0) {
                if (i == 1 && rec.BSDF->reflective()) 
                {
                    return vec3(0., 0., 0.);
                }
                vec3 unit_direction = unit_vector(cur_ray.direction());
                point_light pt_l = point_light(vec3(1., 1., 1.), vec3(0., .8, 0));
                vec3 light_dir;
                double light_dist;
                double pdf;
                vec3 wi;
                vec3 light_radiance = pt_l.sample_light(rec.p, &light_dir, &light_dist, &pdf);
                ray shadow_ray = ray(rec.p, light_dir);
                if (!(*world)->hit(shadow_ray, 0.001f, FLT_MAX, rec)) 
                {
                    double cos = abs(light_dir.z());
                    return cur_attenuation * light_radiance;
                }
            }
            else 
            {
                return vec3(0., 0., 0.);
            }
        }
      }
   return vec3(0.0,0.0,0.0); // exceeded recursion
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, int max_depth, camera **cam, hitable **world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    hit_record rec;
    vec3 col(0,0,0);
    for(int s=0; s < ns; s++) 
    {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u,v);
        r.depth = max_depth;
        col += global_illumination(r, world, &local_rand_state);
    }
    fb[pixel_index] = col/float(ns);
}

__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera, int num_prims, vec3** vec_list) {
    diffuse *red = new diffuse(vec3(1.f, 0.f, 0.f));
    emissive *e = new emissive(vec3(1., 1., 1.), vec3(0., 0., 0.));
    mirror *m = new mirror(vec3(1.f, 1.f, 1.f));
    diffuse *green = new diffuse(vec3(0.f, 1.f, 0.f));
    diffuse *blue = new diffuse(vec3(0.f, 0.f, 1.f));
    diffuse *yellow = new diffuse(vec3(1.f, 1.f, 0.f));
    diffuse *white = new diffuse(vec3(1.f, 1.f, 1.f));
    diffuse *p = new diffuse(vec3(1.f, 0.f, 1.f) * 2.);
    // Image
    auto aspect_ratio = 16.0 / 9.0;
    int image_width = 1900;
    vec3 lookfrom(0,0,5);
    vec3 lookat(0,0,0);
    vec3 vup(0,1,0);
    auto dist_to_focus = 10.0;
    auto aperture = 0.1;
    int image_height = static_cast<int>(image_width / aspect_ratio);
    if (threadIdx.x == 0 && blockIdx.x == 0) 
    {   
        // std::string s("/home/kenny/Documents/Projects/cuda-rt/meshes/obj/box_stack.obj");
        diffuse *white = new diffuse(vec3(1.f, 1.f, 1.f));
        // Check to see if it loaded
        // std::ofstream file("e1Out.txt");
        int count = 0;
        // Create/Open e1Out.txt

        // Copy one of the loaded meshes to be our current mesh
        // for (int j = 0; j < 36; j += 1)
        // {
        //     vec3 v1 = vec_list[j * 3];
        //     vec3 v2 = vec_list[j * 3 + 1];
        //     vec3 v3 = vec_list[j * 3 + 2];
        //     // printf("%f", v1.x());
        //     vec3 v4 = vec3(v1.x(), v1.y(), v1.z());
        //     vec3 v5 = vec3(v2.x(), v2.y(), v2.z());
        //     vec3 v6 = vec3(v3.x(), v3.y(), v3.z());
        //     // printf("%f", v1.x());
        //     *(d_list+j) = new triangle(v1, v2, v3, white);
        //     // // std::cerr << i1 << "\n";
        //     // count += 1;
        // }
        *(d_list+0)   = new sphere(vec3(0,-.6,-.5), 0.3, m);
        // *(d_list+1) = new sphere(vec3(0,0,-.5), 0.1, green);
        // *(d_list+2) = new sphere(vec3(.5,-.6,-.5), 0.2, blue);
        // *(d_list+3) = new sphere(vec3(-.6,-.6,-.5), 0.23, yellow);
        // // *(d_list+4) = new sphere(vec3(0,-10.5,-1), 10, p);
        // // *(d_list+5) = new triangle(vec3(0,-.6,-3), vec3(0, 3, -3), vec3(1, -.5, -3), m);
        // *(d_list+4) = new triangle(vec3(-1,-1,-3), vec3(-1, 1, -3), vec3(1, 1, -3), white);
        // *(d_list+5) = new triangle(vec3(1, 1, -3), vec3(-1,-1,-3), vec3(1, -1, -3), white);
        // // *(d_list+6) = new triangle(vec3(-1,1,-3), vec3(1, 1, -3), vec3(1, 1, 0), white);
        // // *(d_list+7) = new triangle(vec3(-1,1,-3), vec3(-1, 1, 0), vec3(1, 1, 0), white);
        // *(d_list+6) = new triangle(vec3(-1,-1,-3), vec3(1, -1, -3), vec3(1, -1, 0), white);
        // *(d_list+7) = new triangle(vec3(-1,-1,-3), vec3(-1, -1, 0), vec3(1, -1, 0), white);
        // *(d_list+8) = new triangle(vec3(-1,-1,-3), vec3(-1, 1, -3), vec3(-1, 1, 0), white);
        // *(d_list+9) = new triangle(vec3(-1,-1,-3), vec3(-1, -1, 0), vec3(-1, 1, 0), white);
        // *(d_list+10) = new triangle(vec3(1,-1,-3), vec3(1, 1, -3), vec3(1, 1, 0), white);
        // *(d_list+11) = new triangle(vec3(1,-1,-3), vec3(1, -1, 0), vec3(1, 1, 0), white);
        // *(d_list+12) = new triangle(vec3(-1,1,-3), vec3(1, 1, -3), vec3(1, 1, 0), white);
        // *(d_list+13) = new triangle(vec3(-1,1,-3), vec3(-1, 1, 0), vec3(1, 1, 0), white);
        *d_world    = new hitable_list(d_list,1);
        *d_camera   = new camera(lookfrom, lookat, vup, 20, aspect_ratio, aperture, dist_to_focus, 0.0, 1.0);
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera, int num_prims) {
    for (int i = 0; i < 1; i++) 
    {
        delete *(d_list + i);
    }
    delete *d_world;
    delete *d_camera;
}

__host__ int load_mesh(std::string s, std::vector<vec3> vec_list) {
    // Initialize Loader
	objl::Loader Loader;
	// Load .obj File
	bool loadout = Loader.LoadFile(s);
    // diffuse *white = new diffuse(vec3(1.f, 1.f, 1.f));
	// Check to see if it loaded
    // std::ofstream file("e1Out.txt");
    int count = 0;
	// If so continue
	if (loadout)
	{
        // Create/Open e1Out.txt

		// Go through each loaded mesh and out its contents
		for (int i = 0; i < Loader.LoadedMeshes.size(); i++)
		{
			// Copy one of the loaded meshes to be our current mesh
			objl::Mesh curMesh = Loader.LoadedMeshes[i];
            for (int j = 0; j < curMesh.Indices.size(); j += 3)
			{
				int i1 = curMesh.Indices[j];
                int i2 = curMesh.Indices[j + 1];
                int i3 = curMesh.Indices[j + 2];
                // std::cerr << curMesh.Vertices[i1].Position.X << " " << curMesh.Vertices[i1].Position.Y << " " << curMesh.Vertices[i1].Position.Z << "\n";
                std::cerr << count << "\n";
                // std::cerr << "count" << "\n";
                // vec3* v1 = new vec3(curMesh.Vertices[i1].Position.X, curMesh.Vertices[i1].Position.Y, curMesh.Vertices[i1].Position.Z);
                // vec3* v2 = new vec3(curMesh.Vertices[i2].Position.X, curMesh.Vertices[i2].Position.Y, curMesh.Vertices[i2].Position.Z);
                // vec3* v3 = new vec3(curMesh.Vertices[i3].Position.X, curMesh.Vertices[i3].Position.Y, curMesh.Vertices[i3].Position.Z);
                vec_list[count] = vec3(curMesh.Vertices[i1].Position.X, curMesh.Vertices[i1].Position.Y, curMesh.Vertices[i1].Position.Z);
                vec_list[count + 1] = vec3(curMesh.Vertices[i2].Position.X, curMesh.Vertices[i2].Position.Y, curMesh.Vertices[i2].Position.Z);
                vec_list[count + 2] = vec3(curMesh.Vertices[i3].Position.X, curMesh.Vertices[i3].Position.Y, curMesh.Vertices[i3].Position.Z);
                // vec3 v4 = **(vec_list+count);
                // std::cerr << vec_list[count].x() << "\n";
                count += 3;
			}
            // std::cerr << count << "\n";
        }
        return count;
    }
    else
	{
		// Output Error
		std::cerr << "Failed to Load File. May have failed to find it or it was not an .obj file.\n";
        return count;
	}

}

int main() {
    // std::string s("/home/kenny/Documents/Projects/cuda-rt/meshes/obj/box_stack.obj");
	// Load .obj File
    // load_mesh();
    // Initialize Loader
    // vec3** vec_list;
    // checkCudaErrors(hipMalloc((void **)&vec_list, 108*sizeof(vec3 *)));
    // std::cerr << count;
	// Load .obj File
	// bool loadout = Loader.LoadFile("/home/kenny/Documents/Projects/cuda-rt/meshes/obj/box_stack.obj");
	// Check to see if it loaded
    // std::ofstream file("e1Out.txt");
	// If so continue
    int nx = 1200;
    int ny = 600;
    int ns = 50;
    int tx = 16;
    int ty = 16;
    int max_depth = 15;
    int num_prims = 108;
    // thrust::host_vector<vec3> vec_list(108);

    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(vec3);

    // allocate FB
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));

    // make our world of hitables & the camera
    // vec3 **vec_list;
    // checkCudaErrors(hipMalloc((void **)&vec_list, 108*sizeof(vec3 *)));
    std::vector<vec3> vec_list(108);
    objl::Loader Loader;
	// Load .obj File
	bool loadout = Loader.LoadFile("/home/kenny/Documents/Projects/cuda-rt/meshes/obj/box_stack.obj");
    // diffuse *white = new diffuse(vec3(1.f, 1.f, 1.f));
	// Check to see if it loaded
    // std::ofstream file("e1Out.txt");
    int count = 0;
	// If so continue
	if (loadout)
	{
        // Create/Open e1Out.txt

		// Go through each loaded mesh and out its contents
		for (int i = 0; i < Loader.LoadedMeshes.size(); i++)
		{
			// Copy one of the loaded meshes to be our current mesh
			objl::Mesh curMesh = Loader.LoadedMeshes[i];
            for (int j = 0; j < curMesh.Indices.size(); j += 3)
			{
				int i1 = curMesh.Indices[j];
                int i2 = curMesh.Indices[j + 1];
                int i3 = curMesh.Indices[j + 2];
                // std::cerr << curMesh.Vertices[i1].Position.X << " " << curMesh.Vertices[i1].Position.Y << " " << curMesh.Vertices[i1].Position.Z << "\n";
                // std::cerr << count << "\n";
                // std::cerr << "count" << "\n";
                // vec3* v1 = new vec3(curMesh.Vertices[i1].Position.X, curMesh.Vertices[i1].Position.Y, curMesh.Vertices[i1].Position.Z);
                // vec3* v2 = new vec3(curMesh.Vertices[i2].Position.X, curMesh.Vertices[i2].Position.Y, curMesh.Vertices[i2].Position.Z);
                // vec3* v3 = new vec3(curMesh.Vertices[i3].Position.X, curMesh.Vertices[i3].Position.Y, curMesh.Vertices[i3].Position.Z);
                vec_list[count] = vec3(curMesh.Vertices[i1].Position.X, curMesh.Vertices[i1].Position.Y, curMesh.Vertices[i1].Position.Z);
                vec_list[count + 1] = vec3(curMesh.Vertices[i2].Position.X, curMesh.Vertices[i2].Position.Y, curMesh.Vertices[i2].Position.Z);
                vec_list[count + 2] = vec3(curMesh.Vertices[i3].Position.X, curMesh.Vertices[i3].Position.Y, curMesh.Vertices[i3].Position.Z);
                // vec3 v4 = **(vec_list+count);
                // std::cerr << vec_list[count].x() << "\n";
                count += 3;
			}
            // std::cerr << count << "\n";
        }
    }
    vec3 **vec_list_dev;
    checkCudaErrors(hipMalloc((void **)&vec_list_dev, 108*sizeof(vec3 *)));
    hitable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, 36*sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    // thrust::device_ptr<vec3> dev_ptr = thrust::device_pointer_cast(vec_list);
    // load_mesh("/home/kenny/Documents/Projects/cuda-rt/meshes/obj/box_stack.obj", vec_list);
    // thrust::device_vector<vec3> vec_list_dev = vec_list;
    // vec3 * vec_list_ptr = thrust::raw_pointer_cast(vec_list_dev.data());
    // hipMemset(vec_list_ptr, 0, 108 * sizeof(vec3));
    for (int j = 0; j < 36; j+=1) {
        vec3 v1 = vec_list[j*3];
        vec3 v2 = vec_list[j*3+1];
        vec3 v3 = vec_list[j*3+2];
        // std::cerr << "x: " << v1.x() << "\n";
        // std::cerr << j << "\n";
    }
    std::cerr << "loaded mesh" << "\n";
    // for (int j = 0; j < 36; j+=1) {
    //     vec3 v1 = vec_list[j*3];
    //     vec3 v2 = vec_list[j*3+1];
    //     vec3 v3 = vec_list[j*3+2];
    //     *(vec_list_dev+j*3) = &v1;
    //     // *(vec_list_dev+j*3+1) = &vec_list[j*3+1];
    //     // *(vec_list_dev+j*3+2) = &vec_list[j*3+2];
    //     std::cerr << "x: " << v1.x() << "\n";
    // }
    // vec3 *s = &vec_list[0];
    hipMemcpy(vec_list_dev, &vec_list[0], vec_list.size()*sizeof(vec3 *), hipMemcpyHostToDevice);
    create_world<<<1,1>>>(d_list,d_world,d_camera,36, vec_list_dev);
    // for (int j = 0; j < 36; j+=1) {
    //     triangle v1 = **(d_list + j*3);
    //     vec3 v2 = **(d_list + j*3 + 1);
    //     vec3 v3 = **(d_list + j*3 + 2);
    //     std::cerr << "x: " << v1.x() << "\n";
    //     std::cerr << j << "\n";
    // }
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    std::cerr << "created world" << "\n";

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny,  ns, max_depth, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int ir = int(255.99*fb[pixel_index].r());
            int ig = int(255.99*fb[pixel_index].g());
            int ib = int(255.99*fb[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world,d_camera,36);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(vec_list_dev));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));
    // thrust::device_free(vec_list_ptr);

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();
}