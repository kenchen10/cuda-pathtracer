#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <float.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
#include "vec3.h"
#include "ray.h"
#include "bsdf.h"
#include "sphere.h"
#include "hitable.h"
#include "hitable_list.h"
#include "camera.h"
#include "sampler.h"
#include "light.h"
#include "triangle.h"
#include "utils/OBJ_Loader.h"

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
            file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ vec3 global_illumination(const ray& r, hitable **world, hiprandState *local_rand_state) {
   ray cur_ray = r;
   vec3 cur_attenuation = vec3(1.0f, 1.0f, 1.0f);
   unit_sphere_sampler sampler;
   vec3 reflectance = vec3(1., 1., 1.);
   vec3 f = reflectance / HIP_PI_F;
   for(int i = 0; i < r.depth; i++) {
      hit_record rec;
      if ((*world)->hit(cur_ray, 0.001f, FLT_MAX, rec)) {
        vec3 target;
        double pdf;
        vec3 f = rec.BSDF->evaluate(r.direction(), &target, rec.p, rec.normal, &pdf, local_rand_state);
        cur_attenuation *= f;
        cur_ray = ray(rec.p, target-rec.p);
      }
      else {
            if (i > 0) {
                vec3 unit_direction = unit_vector(cur_ray.direction());
                point_light pt_l = point_light(vec3(1., 1., 1.), vec3(0., 1., 0.));
                vec3 light_dir;
                double light_dist;
                double pdf;
                vec3 wi;
                vec3 light_radiance = pt_l.sample_light(rec.p, &light_dir, &light_dist, &pdf);
                ray shadow_ray = ray(rec.p, light_dir);
                if (!(*world)->hit(shadow_ray, 0.001f, FLT_MAX, rec)) {
                    double cos = abs(light_dir.z());
                    return cur_attenuation * light_radiance;
                }
            }
            else {
                return vec3(0., 0., 0.);
            }
        }
      }
   return vec3(0.0,0.0,0.0); // exceeded recursion
}

__global__ void render_init(int max_x, int max_y, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    //Each thread gets same seed, a different sequence number, no offset
    hiprand_init(1984, pixel_index, 0, &rand_state[pixel_index]);
}

__global__ void render(vec3 *fb, int max_x, int max_y, int ns, int max_depth, camera **cam, hitable **world, hiprandState *rand_state) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if((i >= max_x) || (j >= max_y)) return;
    int pixel_index = j*max_x + i;
    hiprandState local_rand_state = rand_state[pixel_index];
    hit_record rec;
    vec3 col(0,0,0);
    for(int s=0; s < ns; s++) {
        float u = float(i + hiprand_uniform(&local_rand_state)) / float(max_x);
        float v = float(j + hiprand_uniform(&local_rand_state)) / float(max_y);
        ray r = (*cam)->get_ray(u,v);
        r.depth = max_depth;
        col += global_illumination(r, world, &local_rand_state);
    }
    fb[pixel_index] = col/float(ns);
}

__global__ void create_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    diffuse *red = new diffuse(vec3(1.f, 0.f, 0.f));
    emissive *e = new emissive(vec3(1., 1., 1.), vec3(0., 0., 0.));
    mirror *m = new mirror(vec3(1.f, 1.f, 1.f));
    diffuse *green = new diffuse(vec3(0.f, 1.f, 0.f));
    diffuse *blue = new diffuse(vec3(0.f, 0.f, 1.f));
    diffuse *yellow = new diffuse(vec3(1.f, 1.f, 0.f));
    diffuse *p = new diffuse(vec3(1.f, 0.f, 1.f));
    // Image
    auto aspect_ratio = 16.0 / 9.0;
    int image_width = 400;
    vec3 lookfrom(0,-.2,5);
    vec3 lookat(0,0,0);
    vec3 vup(0,1,0);
    auto dist_to_focus = 10.0;
    auto aperture = 0.1;
    int image_height = static_cast<int>(image_width / aspect_ratio);
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        *(d_list)   = new sphere(vec3(0,0,-1), 0.3, m);
        *(d_list+1)   = new sphere(vec3(-1,0,-1), 0.1, green);
        *(d_list+2)   = new sphere(vec3(.5,0,-.6), 0.2, blue);
        *(d_list+3)   = new sphere(vec3(-.6,0,-1), 0.23, yellow);
        *(d_list+4) = new sphere(vec3(0,-10.5,-1), 10, p);
        *(d_list+5) = new triangle(vec3(0,-.6,-3), vec3(0, 3, -3), vec3(1, -.5, -3), m);
        *d_world    = new hitable_list(d_list,6);
        *d_camera   = new camera(lookfrom, lookat, vup, 20, aspect_ratio, aperture, dist_to_focus, 0.0, 1.0);
    }
}

__global__ void free_world(hitable **d_list, hitable **d_world, camera **d_camera) {
    delete *(d_list);
    delete *(d_list+1);
    delete *(d_list+2);
    delete *(d_list+3);
    delete *(d_list+4);
    delete *(d_list+5);
    delete *d_world;
    delete *d_camera;
}

int main() {
    objl::Loader loader;
    loader.LoadFile("../meshes/obj/bunny.obj");
    int nx = 1200;
    int ny = 600;
    int ns = 10;
    int tx = 8;
    int ty = 8;
    int max_depth = 15;

    std::cerr << "Rendering a " << nx << "x" << ny << " image with " << ns << " samples per pixel ";
    std::cerr << "in " << tx << "x" << ty << " blocks.\n";

    int num_pixels = nx*ny;
    size_t fb_size = num_pixels*sizeof(vec3);

    // allocate FB
    vec3 *fb;
    checkCudaErrors(hipMallocManaged((void **)&fb, fb_size));

    // allocate random state
    hiprandState *d_rand_state;
    checkCudaErrors(hipMalloc((void **)&d_rand_state, num_pixels*sizeof(hiprandState)));

    // make our world of hitables & the camera
    hitable **d_list;
    checkCudaErrors(hipMalloc((void **)&d_list, 6*sizeof(hitable *)));
    hitable **d_world;
    checkCudaErrors(hipMalloc((void **)&d_world, sizeof(hitable *)));
    camera **d_camera;
    checkCudaErrors(hipMalloc((void **)&d_camera, sizeof(camera *)));
    create_world<<<1,1>>>(d_list,d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    clock_t start, stop;
    start = clock();
    // Render our buffer
    dim3 blocks(nx/tx+1,ny/ty+1);
    dim3 threads(tx,ty);
    render_init<<<blocks, threads>>>(nx, ny, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    render<<<blocks, threads>>>(fb, nx, ny,  ns, max_depth, d_camera, d_world, d_rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());
    stop = clock();
    double timer_seconds = ((double)(stop - start)) / CLOCKS_PER_SEC;
    std::cerr << "took " << timer_seconds << " seconds.\n";

    // Output FB as Image
    std::cout << "P3\n" << nx << " " << ny << "\n255\n";
    for (int j = ny-1; j >= 0; j--) {
        for (int i = 0; i < nx; i++) {
            size_t pixel_index = j*nx + i;
            int ir = int(255.99*fb[pixel_index].r());
            int ig = int(255.99*fb[pixel_index].g());
            int ib = int(255.99*fb[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }

    // clean up
    checkCudaErrors(hipDeviceSynchronize());
    free_world<<<1,1>>>(d_list,d_world,d_camera);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipFree(d_camera));
    checkCudaErrors(hipFree(d_world));
    checkCudaErrors(hipFree(d_list));
    checkCudaErrors(hipFree(d_rand_state));
    checkCudaErrors(hipFree(fb));

    // useful for cuda-memcheck --leak-check full
    hipDeviceReset();
}